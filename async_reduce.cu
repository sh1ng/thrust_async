#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <cassert>

#if __cplusplus >= 201103L
#include <future>
#endif

// This example demonstrates two ways to achieve algorithm invocations that are asynchronous with
// the calling thread.
//
// The first method wraps a call to thrust::reduce inside a __global__ function. Since __global__ function
// launches are asynchronous with the launching thread, this achieves asynchrony. The result of the reduction
// is stored to a pointer to CUDA global memory. The calling thread waits for the result of the reduction to 
// be ready by synchronizing with the CUDA stream on which the __global__ function is launched.
//
// The second method uses the C++11 library function, std::async, to create concurrency. The lambda function
// given to std::async returns the result of thrust::reduce to a std::future. The calling thread can use the
// std::future to wait for the result of the reduction. This method requires a compiler which supports
// C++11-capable language and library constructs.

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce_kernel(Iterator first, Iterator last, T init, BinaryOperation binary_op, Pointer result)
{
  *result = thrust::reduce(thrust::cuda::par, first, last, init, binary_op);
}

int main()
{
  const size_t size = 10;
  hipStream_t *streams = new hipStream_t[size];
  thrust::device_vector<unsigned int> *result = new thrust::device_vector<unsigned int>[size];
  for(size_t i =0; i < size; ++i){
    result[i] = thrust::device_vector<unsigned int>(1, 0);
    hipStream_t s;
    hipStreamCreate(&s);
    streams[i] = s;

  }
  size_t n = 1 << 20;
  

  for(size_t i = 0; i < size; ++i){
    thrust::device_vector<unsigned int> data(n, 1);
    hipStream_t s = streams[i];
    reduce_kernel<<<1,1,0,s>>>(data.begin(), data.end(), 0, thrust::plus<int>(), result[i].data());
  
  }

  for(size_t i =0; i < size; ++i){
    hipStream_t s = streams[i];
    hipStreamSynchronize(s);
  }


  // reset the result
  // result[0] = 0;

#if __cplusplus >= 201103L
  // method 2: use std::async to create asynchrony

  // copy all the algorithm parameters
  auto begin        = data.begin();
  auto end          = data.end();
  unsigned int init = 0;
  auto binary_op    = thrust::plus<unsigned int>();

  // std::async captures the algorithm parameters by value
  // use std::launch::async to ensure the creation of a new thread
  std::future<unsigned int> future_result = std::async(std::launch::async, [=]
  {
    return thrust::reduce(begin, end, init, binary_op);
  });

  // wait on the result and check that it is correct
  assert(future_result.get() == n);
#endif

  return 0;
}

