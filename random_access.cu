#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cassert>
#include <random>
#include <algorithm>
#include <iterator>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>

#define PXL_HOST_LOOPS 64

template <class T1>
__global__ void gather_kernel(const unsigned int *const __restrict__ position,
                              const T1 *const __restrict__ in1, T1 *out1,
                              const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    out1[i] = in1[position[i]];
  }
}

int main(int argc, char **argv) {
  const size_t size_MB = max(atoi(argv[1]), 1) * sizeof(unsigned int);
  const size_t size = size_MB * 1024 * 1024;

  thrust::host_vector<unsigned int> index(size);
  thrust::sequence(index.begin(), index.end());

  std::random_device rd;
  std::mt19937 g(rd());

  std::shuffle(index.begin(), index.end(), g);

  thrust::device_vector<unsigned int> index_d = index;
  thrust::device_vector<unsigned int> data_d = index;
  thrust::device_vector<unsigned int> out_d = index;

  int minGridSize;
  int blockSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                     gather_kernel<unsigned int>, 0, 0);

  int gridSize = (size + blockSize - 1) / blockSize;

  // warm-up
  gather_kernel<<<gridSize, blockSize>>>(
      thrust::raw_pointer_cast(index_d.data()),
      thrust::raw_pointer_cast(data_d.data()),
      thrust::raw_pointer_cast(out_d.data()), size);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord(start);

  for (int ii = 0; ii < PXL_HOST_LOOPS; ii++) {
    gather_kernel<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(index_d.data()),
        thrust::raw_pointer_cast(data_d.data()),
        thrust::raw_pointer_cast(out_d.data()), size);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);

  printf("%8lu, %f, %8.4f\n", size_MB, elapsed / PXL_HOST_LOOPS,
         (1.0 * size_MB * PXL_HOST_LOOPS) / (elapsed * 1024));

  return 0;
}
